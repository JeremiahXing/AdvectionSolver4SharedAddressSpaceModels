#include "hip/hip_runtime.h"
// CUDA parallel 2D advection solver module
// written for COMP4300/8300 Assignment 2, 2021
// v1.0 15 Apr 

// ./testAdvect [-h] [-s] [-g Gx[,Gy]] [-b Bx[,By]] [-o] [-w w] [-d d] M N [r]

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "serAdvect.h" // advection parameters

#define MAX_SHARED_MEMO 49152 // max number of bytes in shared memory on GPU per block 

static int M, N, Gx, Gy, Bx, By; // local store of problem parameters
// static int verbosity;

//sets up parameters above
void initParParams(int M_, int N_, int Gx_, int Gy_, int Bx_, int By_, int verb) {
  M = M_;
  N = N_; 
  Gx = Gx_; 
  Gy = Gy_;  
  Bx = Bx_; 
  By = By_; 
  //verbosity = verb;
} //initParParams()


__host__ __device__
static void N2Coeff(double v, double *cm1, double *c0, double *cp1) {
  double v2 = v/2.0;
  *cm1 = v2*(v+1.0);
  *c0  = 1.0 - v*v;
  *cp1 = v2*(v-1.0);
}

__host__ __device__
void myUpdateAdvectField(int M, int N, double *u, int ldu, double *v, int ldv, double Ux, double Uy) {
  double cim1, ci0, cip1, cjm1, cj0, cjp1;
  N2Coeff(Ux, &cim1, &ci0, &cip1);
  N2Coeff(Uy, &cjm1, &cj0, &cjp1);

  for (int i=0; i < M; i++)
  {
    for (int j=0; j < N; j++)
      {
        V(v,i,j) =
        cim1*(cjm1*V(u,i-1,j-1) + cj0*V(u,i-1,j) + cjp1*V(u,i-1,j+1)) +
        ci0 *(cjm1*V(u,i  ,j-1) + cj0*V(u,i,  j) + cjp1*V(u,i,  j+1)) +
        cip1*(cjm1*V(u,i+1,j-1) + cj0*V(u,i+1,j) + cjp1*V(u,i+1,j+1));
        //printf("update v(%d, %d) = %.2f\n", i, j, V(v,i,j));
      }
  }
} //updateAdvectField() 

__host__ __device__
void myCopyField(int M, int N, double *v, int ldv, double *u, int ldu) {
  for (int i=0; i < M; i++)
    for (int j=0; j < N; j++)
      V(u,i,j) = V(v,i,j);
}

__global__ void updateBoundaryNSKernel(int M, int N, double *u, int ldu) {
  // int j = blockIdx.y * blockDim.y + threadIdx.y;
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;
  int xDim = blockDim.x * gridDim.x;
  int yDim = blockDim.y * gridDim.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int j = x*xDim + y; // map 2d thread pool in to 1d fashion
  
  while ( j < N + 2) {
    // printf("M, %d, N: %d, j: %d\n",M, N, j);
    // printf("(0, %d) = (%d, %d)\n", j, M, j);
    // printf("(%d, %d) = (1, %d)\n", M+1, j, j);
    // printf("\n");
    V(u, 0, j) = V(u, M, j);
    V(u, M+1, j) = V(u, 1, j);
    j += xDim * yDim;
  }
}

__global__ void updateBoundaryEWKernel(int M, int N, double *u, int ldu) {
  int xDim = blockDim.x * gridDim.x;
  int yDim = blockDim.y * gridDim.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int i = x*xDim + y;
  
  while (i < M + 2) {
    // printf("M, %d, N: %d, i: %d\n",M, N, i);
    // printf("(%d, 0) = (%d, %d)\n", i, i, N);
    // printf("(%d, %d) = (%d, 1)\n", i, N+1, i);
    // printf("\n");
    V(u, i, 0) = V(u, i, N);
    V(u, i, N+1) = V(u, i, 1);
    i += xDim * yDim;
  }
}

__global__ void updateAdvectFieldKernel(int M, int N, double *u, int ldu, double *v, int ldv, double Ux, double Uy) {
  // Compute unique thread indices within the grid
  int xDim = blockDim.x * gridDim.x;
  int yDim = blockDim.y * gridDim.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  int M0 = (M / xDim) * x;
  int M_loc = (x < xDim - 1) ? (M / xDim) : (M - M0);

  int N0 = (N / yDim) * y;
  int N_loc = (y < yDim - 1) ? (N / yDim) : (N - N0);

  myUpdateAdvectField(M_loc, N_loc, &V(u, M0+1, N0+1), ldu, &V(v, M0+1, N0+1), ldv, Ux, Uy);
}

__global__ void copyFieldKernel(int M, int N, double *v, int ldu, double *u, int ldv) {
  // Compute unique thread indices within the grid
  int xDim = blockDim.x * gridDim.x;
  int yDim = blockDim.y * gridDim.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  int M0 = (M / xDim) * x;
  int M_loc = (x < xDim - 1) ? (M / xDim) : (M - M0);

  int N0 = (N / yDim) * y;
  int N_loc = (y < yDim - 1) ? (N / yDim) : (N - N0);

  //printf("M0: %d, N0: %d, M_loc: %d, N_loc: %d\n", M0, N0, M_loc, N_loc);
  
  myCopyField(M_loc, N_loc, &V(v, M0+1, N0+1), ldu, &V(u, M0+1, N0+1), ldv);
}

__global__ void updateAdvectFieldOpt1(int M, int N, double *u, int ldu, double *v, int ldv, double cim1, double ci0, double cip1, double cjm1, double cj0, double cjp1) {
  extern __shared__ double s[];
  int lds = blockDim.x + 2;

  int si = threadIdx.x + 1;
  int sj = threadIdx.y + 1;

  int ui = blockIdx.x * blockDim.x + si;
  while (ui <= M){
    int uj = blockIdx.y * blockDim.y + sj;
    while (uj <= N){
      // printf("ui: %d, uj: %d\n", ui, uj);
      V(s, si, sj) = V(u, ui, uj);
      // __syncthreads();
      // update shared memo boundary
      if (si == 1) {
        V(s, si-1, sj) = V(u, ui-1, uj);
        if (sj == 1) {
          V(s, si-1, sj-1) = V(u, ui-1, uj-1);
        }
        if (sj == blockDim.y || uj == N) {
          V(s, si-1, sj+1) = V(u, ui-1, uj+1);
        }
      } 
      if (si == blockDim.x || ui == M) {
        V(s, si+1, sj) = V(u, ui+1, uj);
        if (sj == 1) {
          V(s, si+1, sj-1) = V(u, ui+1, uj-1);
        } 
        if (sj == blockDim.y || uj == N) {
          V(s, si+1, sj+1) = V(u, ui+1, uj+1);
        } 
      }
      //__syncthreads();
      if (sj == 1) {
        V(s, si, sj-1) = V(u, ui, uj-1);
      } 
      if (sj == blockDim.y || uj == N) {
        V(s,si, sj+1) = V(u, ui, uj+1);
      }
      __syncthreads();
      // update v
      
      V(v, ui ,uj) =
          cim1*(cjm1*V(s,si-1,sj-1) + cj0*V(s,si-1,sj) + cjp1*V(s,si-1,sj+1)) +
          ci0 *(cjm1*V(s,si  ,sj-1) + cj0*V(s,si,  sj) + cjp1*V(s,si,  sj+1)) +
          cip1*(cjm1*V(s,si+1,sj-1) + cj0*V(s,si+1,sj) + cjp1*V(s,si+1,sj+1));
      __syncthreads();
      uj += blockDim.y * gridDim.y;
    }
    ui += blockDim.x * gridDim.x;
  }
}

__global__ void updateAdvectFieldOpt2(int M, int N, double *u, int ldu, double *v, int ldv, double cim1, double ci0, double cip1, double cjm1, double cj0, double cjp1) {
  // Compute unique thread indices within the grid
  int xDim = blockDim.x * gridDim.x;
  int yDim = blockDim.y * gridDim.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  int M0 = (M / xDim) * x;
  int M_loc = (x < xDim - 1) ? (M / xDim) : (M - M0);

  int N0 = (N / yDim) * y;
  int N_loc = (y < yDim - 1) ? (N / yDim) : (N - N0);
  //myUpdateAdvectField(M_loc, N_loc, &V(u, M0+1, N0+1), ldu, &V(v, M0+1, N0+1), ldv, Ux, Uy);
  u = &V(u, M0+1, N0+1);
  v = &V(v, M0+1, N0+1);
  for (int i=0; i < M_loc; i++)
  {
    for (int j=0; j < N_loc; j++)
      {
        V(v,i,j) =
        cim1*(cjm1*V(u,i-1,j-1) + cj0*V(u,i-1,j) + cjp1*V(u,i-1,j+1)) +
        ci0 *(cjm1*V(u,i  ,j-1) + cj0*V(u,i,  j) + cjp1*V(u,i,  j+1)) +
        cip1*(cjm1*V(u,i+1,j-1) + cj0*V(u,i+1,j) + cjp1*V(u,i+1,j+1));
        //printf("update v(%d, %d) = %.2f\n", i, j, V(v,i,j));
      }
  }
}

// evolve advection over reps timesteps, with (u,ldu) containing the field
// parallel (2D decomposition) variant
void cuda2DAdvect(int reps, double *u, int ldu) {
  double Ux = Velx * dt / deltax;
  double Uy = Vely * dt / deltay;
  int ldv = N + 2;
  double *v;
  HANDLE_ERROR( hipMalloc(&v, ldv*(M+2)*sizeof(double)) );

  dim3 block(Bx, By);
  dim3 grid(Gx, Gy);

  for (int r = 0; r < reps; r++) {
    //test block
    // updateBoundaryNS <<<1,1>>> (N, M, u, ldu);
    // updateBoundaryEW <<<1,1>>> (M, N, u, ldu);
    
    updateBoundaryNSKernel<<<grid, block>>>(M, N, u, ldu); 
    updateBoundaryEWKernel<<<grid, block>>>(M, N, u, ldu); 

    // test block
    // updateAdvectFieldK <<<1,1>>> (M, N, &V(u,1,1), ldu, &V(v,1,1), ldv, Ux, Uy);
    // copyFieldK <<<1,1>>> (M, N, &V(v,1,1), ldv, &V(u,1,1), ldu);

    updateAdvectFieldKernel<<<grid, block>>>(M, N, u, ldu, v, ldv, Ux, Uy); 
    copyFieldKernel <<<grid, block>>> (M, N, v, ldv, u, ldu); 
  } //for(r...)

  HANDLE_ERROR( hipFree(v) );
} //cuda2DAdvect()



// ... optimized parallel variant
void cudaOptAdvect(int reps, double *u, int ldu, int w) {
  // if (M > Gx*Bx)
  //   if (M % (M / (Gx*Bx)) != 0 ){
  //     printf("Please set reasonable Gx and Bx, Gx*Bx is expected be a factor of M.\n");
  //     exit(0);
  //   }
  // if (N > Gy*By) 
  //   if (N % (N / (Gy*By)) != 0 ){
  //     printf("Please set reasonable Gy and By, Gy*By is expected to be a factor of N.\n");
  //     exit(0);
  //   }
  double Ux = Velx * dt / deltax;
  double Uy = Vely * dt / deltay;
  double cim1, ci0, cip1, cjm1, cj0, cjp1;
  N2Coeff(Ux, &cim1, &ci0, &cip1);
  N2Coeff(Uy, &cjm1, &cj0, &cjp1);

  int ldv = N + 2;
  double *v;
  HANDLE_ERROR( hipMalloc(&v, ldv*(M+2)*sizeof(double)) );

  dim3 block(Bx, By);
  dim3 grid(Gx, Gy);

  for (int r = 0; r < reps; r++) {
    updateBoundaryNSKernel<<<grid, block>>>(M, N, u, ldu); 
    updateBoundaryEWKernel<<<grid, block>>>(M, N, u, ldu);
    bool isOpt1 = true;
    if (M > Gx*Bx){
      if (M % (M / (Gx*Bx)) != 0 ){
        isOpt1 = false;
      }
    }
    if (N > Gy*By){
      if (N % (N / (Gy*By)) != 0 ){
        isOpt1 = false;
      }
    } 
    if (isOpt1){
      size_t sharedMemSize = (Bx+2) * (By+2) * sizeof(double);
      //assert(sharedMemSize <= MAX_SHARED_MEMO);
      if (sharedMemSize > MAX_SHARED_MEMO) {
        printf("sharedMemo overflow with requested sharedMemSize: %lu, please try large grid and block size\n", sharedMemSize);
        exit(0);
      }
      updateAdvectFieldOpt1<<<grid, block, sharedMemSize>>>(M, N, u, ldu, v, ldv, cim1, ci0, cip1, cjm1, cj0, cjp1);
    } else {
      updateAdvectFieldOpt2<<<grid, block>>>(M, N, u, ldu, v, ldv, cim1, ci0, cip1, cjm1, cj0, cjp1);
    }
    hipDeviceSynchronize();
    double *tmp = u;
    u = v;
    v = tmp;
    //HANDLE_ERROR( hipMemcpy(u, v, ldv*(M+2)*sizeof(double), hipMemcpyDeviceToDevice) );
  } //for(r...)
  if (reps % 2 == 1) {
    double *tmp = u;
    u = v;
    v = tmp;
    HANDLE_ERROR( hipMemcpy(u, v, ldv*(M+2)*sizeof(double), hipMemcpyDeviceToDevice) );
    //copyFieldKernel <<<grid, block>>> (M, N, v, ldv, u, ldu);
  }
  HANDLE_ERROR( hipFree(v) );
} //cudaOptAdvect()
